#include "hip/hip_runtime.h"
//main.cpp
#include "vose.h"
#include <cmath>
#include <iostream>
#include <algorithm>
#include <cstdio>

using namespace std;

# define PI 3.14159265358979323846

__device__ double potential(double*);
__device__ double potential_osc(double*);
__device__ double potential3(double*);
__device__ double kinetic(double*);
__device__ double Hamiltonian(double*, double*, unsigned int, double);
__global__ void update_state(double*, double*, unsigned int, double, double);
void write_state(FILE*, double, double*, double*, unsigned int);

const double length = 10;
double mass = 0.005;

int main(int argc, char *argv[])
{
    auto seed = chrono::high_resolution_clock::now().time_since_epoch().count();
    mt19937 mt(seed); // Seeds Mersenne Twister with Device RNG
    
    bool random_sampling = false;
    
    // Define the density array for position
    unsigned int grid_size = 10000; //number of grid points
    unsigned int particle_number = 64;
    
    double *positions;
    hipMallocManaged(&positions, particle_number*sizeof(double));
    
    // Random sampling
    if (random_sampling)
    {
        double *density = new double[grid_size];
        double arg_scale = 2*PI/grid_size;
        double norm = 0;
        for (unsigned int i = 0; i<grid_size; i++)
        {
            density[i] = 0.5*sin(arg_scale*i)+1;
            norm += density[i];
        }
    
        //Normalize position density
        for (unsigned int i = 0; i<grid_size; i++)
        {
            density[i] /= norm;
        }
        
        // Sample locations
        vose *pos_sampler = new vose(density, grid_size, mt);
    
        for (unsigned int i=0; i<particle_number; i++)
        {   
            positions[i] = pos_sampler->alias_method()*length/grid_size;
        }

        //Sort locations (just so IC is nicer)
        sort(positions, positions+particle_number);
    }
    
    // Even spacing
    else
    {
        double spacing = length/(particle_number+1);
        for (unsigned int i=0; i<particle_number; i++)
        {
            positions[i] = spacing*(i+1);
        }
    }
    
    
    // Define momentum density
    double *mom_density = new double[grid_size];
    double Temperature = 100;
    double arg_scale = -1/(2*mass*Temperature);
    
    double norm = 0;
    for (unsigned int i = -floor(grid_size/2); i<floor(grid_size/2); i++)
    {
        mom_density[i] = exp(arg_scale*i*i);
        norm += mom_density[i];
    }
    
    //Normalize momentum density
    for (unsigned int i = 0; i<grid_size; i++)
    {
        mom_density[i] /= norm;
    }
    
    vose *mom_sampler = new vose(mom_density, grid_size, mt);
    
    double *momenta;
    hipMallocManaged(&momenta, particle_number*sizeof(double));
    
    double mom_scale = 0.0005;
    unsigned int j;
    for (unsigned int i=0; i<particle_number; i++)
    {   
        //j = mom_sampler->alias_method(); //momentum index
        //momenta[i] = (j-float(grid_size)/2)*mom_scale; //convert j to momentum
        momenta[i] = 0;
    }
    
    /*--------------------------------------------*/
    
	//Create output file
	FILE *output_file;
	output_file = fopen("output.tsv", "w");
	fprintf(output_file, "Time\tPositions\tMomenta\n");
    
    // Hamiltonian solution part
    double dt = 0.01;
    double max_time = 1200;
    
    int counter = 0;
    for (double t = 0; t<max_time; t+=dt)
    {
        cout << t << endl;
        update_state<<<1,256>>>(positions, momenta, particle_number, dt, t);
        counter++;
        if (false)//(counter % 10 == 0)
        {
            // Wait for GPU to finish before accessing on host
            // Otherwise you get a Bus error: 10
            hipDeviceSynchronize();
            
            write_state(output_file, t, positions, momenta, particle_number);
        }
    }
    
    hipFree(positions);
    hipFree(momenta);
    
    return 0;
}

__device__
double potential(double *positions, unsigned int particle_number)
{
    double k = 0.1;
    double potential_energy = 0;
    double scale = k/2;
    double spacing = length/particle_number;
    
    for (unsigned int i = 0; i<=particle_number; i++)
    {
        if (i==0)
            potential_energy += scale*pow(positions[0]-spacing,2);
        else if (i==particle_number)
            potential_energy += scale*pow(length-positions[i-1]-spacing,2);
        else
            potential_energy += scale*pow(positions[i]-positions[i-1]-spacing,2);
    }
    
    return potential_energy;
}

__device__
double potential_osc(double *positions, unsigned int particle_number, double t)
{
    double k = 0.1;
    double potential_energy = 0;
    double scale = k/2;
    double spacing = length/particle_number;
    
    for (unsigned int i = 0; i<=particle_number; i++)
    {
        if (i==0)
        {
            double driver = 10*sin(2*PI*t/10);
            potential_energy += scale*pow(positions[0]-spacing-driver,2);
        }
        else if (i==particle_number)
            potential_energy += scale*pow(length-positions[i-1]-spacing,2);
        else
            potential_energy += scale*pow(positions[i]-positions[i-1]-spacing,2);
    }
    
    return potential_energy;
}

__device__
double potential3(double *positions, unsigned int particle_number)
{
    double k = 0.1;
    double potential_energy = 0;
    double scale = k/2;
    double spacing = length/particle_number;
    
    for (unsigned int i = 0; i<=particle_number; i++)
    {
        if (i==0)
            potential_energy += scale*fabs(pow(positions[0]-spacing,3));
        else if (i==particle_number)
            potential_energy += scale*fabs(pow(length-positions[i-1]-spacing,3));
        else
            potential_energy += scale*fabs(pow(positions[i]-positions[i-1]-spacing,3));
    }
    
    return potential_energy;
}

__device__
double kinetic(double *momenta, unsigned int particle_number)
{
    double kinetic_energy = 0;
    double mass = 10;
    double scale = 1/(2*mass);
    
    for (unsigned int i = 0; i<particle_number; i++)
    {
        kinetic_energy += scale*pow(momenta[i],2);
    }
    
    return kinetic_energy;
}

__device__
double Hamiltonian(double *positions,double *momenta,unsigned int particle_number, double t)
{
    return potential_osc(positions,particle_number, t) + kinetic(momenta,particle_number);
}

//Gauss-Seidel I think
__global__
void update_state(double *pos, double *mom, unsigned int part_num, double dt, double t)
{
    double dx = 0.001, dp = 0.001; //finite differences
    double H_pplus, H_pminus, H_xplus, H_xminus;
    double p_grad, x_grad;
    
    int index = threadIdx.x;
    int stride = blockDim.x;
    
    for (int i = index; i<part_num; i += stride)
    {
        //Calculate phase space gradient
        pos[i] += dx;
        H_xplus = Hamiltonian(pos, mom, part_num, t);
        pos[i] -= 2*dx;
        H_xminus = Hamiltonian(pos, mom, part_num, t);
        pos[i] += dx;
        x_grad = (H_xplus-H_xminus)/(2*dx);
        
        //Update momentum
        mom[i] -= dt*x_grad;
        
        mom[i] += dp;
        H_pplus = Hamiltonian(pos, mom, part_num, t);
        mom[i] -= 2*dp;
        H_pminus = Hamiltonian(pos, mom, part_num, t);
        mom[i] += dp;
        p_grad = (H_pplus-H_pminus)/(2*dp);
        
        //Update position
        pos[i] += dt*p_grad;
        
    }
    //cout << Hamiltonian(pos, mom, part_num) << endl;
}

void write_state(FILE *fp, double t, double *pos, double *mom, unsigned int part_num)
{
    cout << "check 1\n";
    
    //Write time
    fprintf(fp,"%f\t",t);
    
    cout << "check 2\n";
    
    //Write positions
    for (unsigned int i = 0; i<part_num; i++)
    {
        fprintf(fp,"%f\t",pos[i]);
    }
    
    cout << "check 3\n";
    
    //Write momenta
    for (unsigned int i = 0; i<part_num; i++)
    {
        fprintf(fp,"%f",mom[i]);
        if (i<part_num-1)
            fprintf(fp,"\t");
    }
    
    cout << "check 4\n";
    
    fprintf(fp,"\n");
    
}